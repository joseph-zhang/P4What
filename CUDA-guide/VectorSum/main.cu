#include "hip/hip_runtime.h"
#include "vectorSum.cuh"
#include <iostream>
#include <cmath>
#include <assert.h>
#include <hip/hip_runtime.h>

using namespace std;

int main(int argc, char** argv) {
  const uint per_block_thread_count = 1024;
  const uint max_block_count = 65535;

  const uint array_size = (uint)1e7;

  float* a = new float[array_size];
  float* b = new float[array_size];
  float* c = new float[array_size];
  float* time = new float;

  for(uint i = 0;i < array_size; i++) {
    a[i] = i;
    b[i] = array_size - i;
  }

  float* dev_a;
  float* dev_b;
  float* dev_c;
  hipMalloc((void**) &dev_a, array_size * sizeof(float));
  hipMalloc((void**) &dev_b, array_size * sizeof(float));
  hipMalloc((void**) &dev_c, array_size * sizeof(float));

  hipMemcpy(dev_a, a, array_size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, array_size * sizeof(float), hipMemcpyHostToDevice);

  uint block_count = min(max_block_count, (uint) ceil(array_size / (float) per_block_thread_count));
  cudaCallAddVectorKernel(block_count,
                          per_block_thread_count,
                          dev_a,
                          dev_b,
                          dev_c,
                          array_size,
                          time);

  hipMemcpy(c, dev_c, array_size * sizeof(float), hipMemcpyDeviceToHost);

  for(uint i = 0; i < array_size; i++) {
    assert(c[i] == array_size);
  }
  cout << "GPU runtime: " << *time << " ms" << endl;
  cout << "program check complete. (PASS)" << endl;

  delete[] a;
  delete[] b;
  delete[] c;
  delete time;
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  return 0;
}