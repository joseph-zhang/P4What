#include "hip/hip_runtime.h"
#include "vectorSum.cuh"

__global__
void cudaAddVectorKernel(const float *a,
                         const float* b,
                         float* c,
                         const uint size) {
  uint thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  while (thread_idx < size) {
    c[thread_idx] = a[thread_idx] + b[thread_idx];
    thread_idx += blockDim.x * gridDim.x;
  }
}

void cudaCallAddVectorKernel(const uint block_count,
                             const uint per_block_thread_count,
                             const float* a,
                             const float* b,
                             float* c,
                             const uint size,
                             float* time) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  cudaAddVectorKernel<<<block_count, per_block_thread_count>>>(a, b, c, size);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  *time =  elapsedTime;
}